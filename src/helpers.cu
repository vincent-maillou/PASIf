#include "hip/hip_runtime.h"
/**
 * @file helpers.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2022-11-30
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include "helpers.cuh"



/****************************************************
 *              COO Tensor
 ****************************************************/

  /**
  * @brief Construct a new COOMatrix::COOMatrix object
  * 
  * @param denseMatrix 
  * @param scaleMatrix 
  */
    COOMatrix::COOMatrix(std::vector< matrix > & denseMatrix, std::vector< matrix > & scaleMatrix) :
        n(0),
        alpha(1),
        beta(1){
      // Set device pointer to nullprt
      d_val = nullptr;
      d_row = nullptr;
      d_col = nullptr;

      d_buffer = nullptr;
      bufferSize = 0;

      d_alpha = nullptr;
      d_beta = nullptr;

      // Fill the COO matrix with the values of the vector of dense matrix
      for(size_t k(0); k<denseMatrix.size(); ++k){

        for(size_t i(0); i<denseMatrix[k].size(); ++i){
          for(size_t j(0); j<denseMatrix[k][i].size(); ++j){
            if(std::abs(denseMatrix[k][i][j]) > reel_eps && std::abs(scaleMatrix[k][i][j]) > reel_eps){
              row.push_back(i+n);
              col.push_back(j+n);
              val.push_back(scaleMatrix[k][i][j]*denseMatrix[k][i][j]);
            }
          }
        }
        n += denseMatrix[k].size();

      }
      nzz = val.size();

    }

  /**
  * @brief Destroy the COOMatrix::COOMatrix object
  * 
  */
    COOMatrix::~COOMatrix(){
      if(d_val != nullptr){
        CHECK_CUDA( hipFree(d_val) );
      }
      if(d_row != nullptr){
        CHECK_CUDA( hipFree(d_row) );
      }
      if(d_col != nullptr){
        CHECK_CUDA( hipFree(d_col) );
      }
      if(d_buffer != nullptr){
        CHECK_CUDA( hipFree(d_buffer) );
      }
      if(d_alpha != nullptr){
        CHECK_CUDA( hipFree(d_alpha) );
      }
      if(d_beta != nullptr){
        CHECK_CUDA( hipFree(d_beta) );
      }
    }

  /**
   * @brief Extend the COO Matrix by appending n times the same matrix
   * 
   * @param n 
   */
    uint COOMatrix::ExtendTheSystem(uint nTimes){
      if(nTimes == 0){
        return n;
      }
      
      for(uint i(0); i<nTimes; ++i){
        for(uint j(0); j<nzz; ++j){
          row.push_back(row[j]+(i+1)*n);
          col.push_back(col[j]+(i+1)*n);
          val.push_back(val[j]);
        }
      }
      n += nTimes*n;
      nzz = val.size();

      return n;
    }

  /**
   * @brief Construct a new COOMatrix::AllocateOnGPU object
   * 
   */
    void COOMatrix::AllocateOnGPU(hipsparseHandle_t & handle, hipsparseDnVecDescr_t & vecX, hipsparseDnVecDescr_t & vecY){
      // Allocate memory on the device
      CHECK_CUDA( hipMalloc((void**)&d_row, nzz*sizeof(uint)) );
      CHECK_CUDA( hipMalloc((void**)&d_col, nzz*sizeof(uint)) );
      CHECK_CUDA( hipMalloc((void**)&d_val, nzz*sizeof(reel)) );

      // Copy the data to the device
      CHECK_CUDA( hipMemcpy(d_row, row.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipMemcpy(d_col, col.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipMemcpy(d_val, val.data(), nzz*sizeof(reel), hipMemcpyHostToDevice) );

      // Create the sparse matrix descriptor and allocate the needed buffer
      CHECK_CUSPARSE( hipsparseCreateCoo(&sparseMat_desc, n, n, 
                                        nzz, d_row, d_col, d_val, 
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
      
      CHECK_CUDA( hipMalloc((void**)&d_alpha, sizeof(reel)) );
      CHECK_CUDA( hipMalloc((void**)&d_beta, sizeof(reel)) );

      CHECK_CUSPARSE( hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                              &d_alpha, sparseMat_desc, vecX, &d_beta, vecY, HIP_R_32F, 
                                              HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )

      CHECK_CUDA( hipMalloc((void**)&d_buffer, bufferSize) );
    }

    size_t COOMatrix::memFootprint(){
      // Return the number of bytes needed to store this element on the GPU
      size_t memFootprint;

      memFootprint = bufferSize + 2*nzz*sizeof(uint) + nzz*sizeof(reel); 

      return memFootprint;
    }
  
    std::ostream& COOMatrix::print(std::ostream& out) const{
      // Print the sparse COO matrix in a readable format
      if(nzz == 0){
        out << "Empty matrix" << std::endl;
        return out;
      }

      size_t k(0);
      for(size_t i(0); i<n; ++i){
        for(size_t j(0); j<n; ++j){
          if(col[k] == j && row[k] == i){
            out << val[k] << " ";
            ++k;
          }
          else{
            out << "_ ";
          }
        } 
        out << std::endl;
      }
      out << std::endl;
      
      return out;
    }

    std::ostream& operator<<(std::ostream& out, COOMatrix const& mat){
      return mat.print(out);
    }   



/****************************************************
 *              COO Tensor
 ****************************************************/

  /**
  * @brief Construct a new COOTensor::COOTensor object
  * 
  * @param denseTensor 
  * @param scaleMatrix 
  */
    COOTensor::COOTensor(std::vector< tensor > & denseTensor, std::vector< matrix > & scaleMatrix) : n(0){
      // Set device pointer to nullprt
      d_val = nullptr;
      d_row = nullptr;
      d_col = nullptr;
      d_slice = nullptr;


      // Fill the COO Tensor with the values of the vector of dense tensor
      for(size_t l(0); l<denseTensor.size(); ++l){

        for(size_t k(0); k<denseTensor[l].size(); ++k){
          for(size_t i(0); i<denseTensor[l][k].size(); ++i){
            for(size_t j(0); j<denseTensor[l][k][i].size(); ++j){
              if(denseTensor[l][k][i][j] != 0){
                row.push_back(i+n);
                col.push_back(j+n);
                slice.push_back(k+n);
                val.push_back(scaleMatrix[l][k][k] * denseTensor[l][k][i][j]);
              }
            }
          }
        }
        n += denseTensor[l].size();
      
      }
      nzz = val.size();

    }

  /**
  * @brief Destroy the COOTensor::COOTensor object
  * 
  */
    COOTensor::~COOTensor(){
      if(d_val != nullptr){
        CHECK_CUDA( hipFree(d_val) );
      }
      if(d_row != nullptr){
        CHECK_CUDA( hipFree(d_row) );
      }
      if(d_col != nullptr){
        CHECK_CUDA( hipFree(d_col) );
      }
      if(d_slice != nullptr){
        CHECK_CUDA( hipFree(d_slice) );
      }
    }

  /**
   * @brief Extend the COO Tensor by appending n times the same tensor
   * 
   * @param nTimes 
   */
    uint COOTensor::ExtendTheSystem(uint nTimes){
      if(nTimes == 0){
        return n;
      }

      for(uint i(0); i<nTimes; ++i){
        for(uint j(0); j<nzz; ++j){
          row.push_back(row[j]+(i+1)*n);
          col.push_back(col[j]+(i+1)*n);
          slice.push_back(slice[j]+(i+1)*n);
          val.push_back(val[j]);
        }
      }
      n += nTimes*n;
      nzz = val.size();

      return n;
    }

  /**
   * @brief Construct a new COOTensor::AllocateOnGPU object
   * 
   */
    void COOTensor::AllocateOnGPU(){
      // Allocate memory on the device
      CHECK_CUDA( hipMalloc((void**)&d_row, nzz*sizeof(uint)) );
      CHECK_CUDA( hipMalloc((void**)&d_col, nzz*sizeof(uint)) );
      CHECK_CUDA( hipMalloc((void**)&d_slice, nzz*sizeof(uint)) );
      CHECK_CUDA( hipMalloc((void**)&d_val, nzz*sizeof(reel)) );

      // Copy the data to the device
      CHECK_CUDA( hipMemcpy(d_row, row.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipMemcpy(d_col, col.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipMemcpy(d_slice, slice.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipMemcpy(d_val, val.data(), nzz*sizeof(reel), hipMemcpyHostToDevice) );
    }

    size_t COOTensor::memFootprint(){
      // Return the number of bytes needed to store this element on the GPU
      size_t memFootprint;

      memFootprint = 3*nzz*sizeof(uint) + nzz*sizeof(reel); 

      return memFootprint;
    }

    std::ostream& COOTensor::print(std::ostream& out) const{
      if(nzz == 0){
        out << "Empty COO Tensor" << std::endl;
        return out;
      }


      /* // For debug print the array of the COO Tensor
      std::cout << "val: ";
      for(size_t i(0); i<nzz; ++i){
        std::cout << val[i] << " ";
      }
      std::cout << std::endl;
      std::cout << "row: ";
      for(size_t i(0); i<nzz; ++i){
        std::cout << row[i] << " ";
      }
      std::cout << std::endl;
      std::cout << "col: ";
      for(size_t i(0); i<nzz; ++i){
        std::cout << col[i] << " ";
      }
      std::cout << std::endl;
      std::cout << "slice: ";
      for(size_t i(0); i<nzz; ++i){
        std::cout << slice[i] << " ";
      }
      std::cout << std::endl; */


      size_t p(0);
      for(size_t m(0); m<n; ++m){
        size_t k(p);
        for(size_t j(0); j<n; ++j){
          for(size_t i(0); i<n; ++i){
            if(row[k] == j && col[k] == i && slice[k] == m){
              out << val[k] << " ";
              ++k;
              ++p;
            }
            else{
              out << "_ ";
            }
          }
          out << std::endl;
        }
        out << std::endl;
      }
      out << std::endl;
      
      return out;
    }

    std::ostream& operator<<(std::ostream& out, COOTensor const& tensor_){
      return tensor_.print(out);
    }



/****************************************************
 *              COO Vector
 ****************************************************/
  /** COOVector::COOVector()
    * @brief Construct a new COOVector::COOVector object
    * 
    * @param denseVector 
    */  
    COOVector::COOVector(std::vector< std::vector<reel> > & denseVector, std::vector< matrix > & scaleMatrix) : n(0) {
      d_val = nullptr;
      d_indice = nullptr;

      for(size_t i(0); i<denseVector.size(); ++i){
        for(size_t j(0); j<denseVector[i].size(); ++j){
          if(std::abs(denseVector[i][j]) > reel_eps){
            indice.push_back(j+n);
            // std::abs because the forces applied should be positive
            val.push_back(std::abs(scaleMatrix[i][j][j])*denseVector[i][j]);
          }
        }
        n += denseVector[i].size();

      }
      nzz = val.size();

    }


  /** COOVector::COOVector()
    * @brief Construct a new COOVector::COOVector object
    * 
    * @param denseVector 
    */  
    COOVector::~COOVector(){
      if(d_val != nullptr){
        CHECK_CUDA( hipFree(d_val) );
      }
      if(d_indice != nullptr){
        CHECK_CUDA( hipFree(d_indice) );
      }
    }

  uint COOVector::ExtendTheSystem(uint nTimes){
    if(nTimes == 0){
      return n;
    }

    for(uint i(0); i<nTimes; ++i){
      for(uint j(0); j<nzz; ++j){
        indice.push_back(indice[j]+(i+1)*n);
        val.push_back(val[j]);
      }
    }
    n += nTimes*n;
    nzz = val.size();

    return n;
  }

  void COOVector::AllocateOnGPU(){
    // Allocate memory on the device
    CHECK_CUDA( hipMalloc((void**)&d_indice, nzz*sizeof(uint)) );
    CHECK_CUDA( hipMalloc((void**)&d_val, nzz*sizeof(reel)) );

    // Copy the data to the device
    CHECK_CUDA( hipMemcpy(d_indice, indice.data(), nzz*sizeof(uint), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(d_val, val.data(), nzz*sizeof(reel), hipMemcpyHostToDevice) );

    // Create the sparse vector descriptor
    CHECK_CUSPARSE( hipsparseCreateSpVec(&sparseVec_desc, n, nzz, &d_indice, &d_val,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
  }

  size_t COOVector::memFootprint(){
    // Return the number of bytes needed to store this element on the GPU
    size_t memFootprint;

    memFootprint = nzz*sizeof(uint) + nzz*sizeof(reel); 

    return memFootprint;
  }

  std::ostream& COOVector::print(std::ostream& out) const{
    if(nzz == 0){
      out << "Empty COO Vector" << std::endl;
      return out;
    }

    out << "val: ";
    size_t p(0);
    for(size_t i(0); i<n; ++i){
      if(indice[p] == i){
        out << val[p] << " ";
        ++p;
      }
      else{
        out << "_ ";
      }
    }
    out << std::endl;

    out << "ind: ";
    p = 0;
    for(size_t i(0); i<n; ++i){
      if(indice[p] == i){
        out << indice[p] << " ";
        ++p;
      }
      else{
        out << "_ ";
      }
    }
    out << std::endl;

    return out;
  }

  std::ostream& operator<<(std::ostream& out, COOVector const& vector_){
    return vector_.print(out);
  }



/****************************************************
 *              Utilities
 ****************************************************/

  /**
  * @brief 
  * 
  * @param vectMat 
  * @param scaleFactor 
  */
    void invertMatrix(std::vector< matrix > & vectMat, float scaleFactor){
      // For now just return the invert of each element multiplied by the scale factor
      for(size_t k(0); k<vectMat.size(); ++k){
        for(size_t i(0); i<vectMat[k].size(); ++i){
          for(size_t j(0); j<vectMat[k][i].size(); ++j){
            if(std::abs(vectMat[k][i][j]) > reel_eps){
              vectMat[k][i][j] = scaleFactor/vectMat[k][i][j];
            }
            else{
              vectMat[k][i][j] = 0.;
            }
          }
        }
      }
    }


  std::ostream& operator<<(std::ostream& out, matrix const& mat){
    // Print the row-major dense matrix in the output stream
    for(size_t i(0); i<mat.size(); ++i){
      for(size_t j(0); j<mat[i].size(); ++j){
        out << mat[i][j] << " ";
      }
      out << std::endl;
    }

    return out;
  }

  void printVector(std::vector<reel> & vec){
    std::cout << vec << std::endl;
  }


  template <typename T>
  std::ostream& operator<<(std::ostream& out, std::vector<T> const& vec){
    for(size_t i(0); i<vec.size(); ++i){
      out << vec[i] << " ";
    }
    out << std::endl;
    return out;
  }      

  uint extendTheVector(std::vector<reel> & vec, uint nTimes){
    uint n(vec.size());
    for(uint i(0); i<nTimes; ++i){
      for(uint j(0); j<n; ++j){
        vec.push_back(vec[j]);
      }
    }
    return vec.size();
  }



  