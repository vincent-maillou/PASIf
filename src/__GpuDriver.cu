#include "hip/hip_runtime.h"
/**
 * @file __GpuDriver.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2022-11-29
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include "__GpuDriver.cuh"



/****************************************************
 *              Public functions                   *
 ****************************************************/
  /** __GpuDriver::__GpuDriver()
    * @brief Construct a new gpudriver::  gpudriver object
    * 
    * @param excitationSet_ 
    * @param sampleRate_ 
    */
    __GpuDriver::__GpuDriver(std::vector<std::vector<double>> excitationSet_, 
                             uint sampleRate_,
                             uint numsteps_) : 
        // Simulation
        n_dofs(0),
        baseNumsteps(numsteps_),
        //simNumsteps(numsteps_),

        // Interpolation
        interpolationNumberOfPoints(0),
        interpolationWindowSize(0),

        // Computation parameters
        nStreams(1),
        intraStrmParallelism(1),
        numberOfSimulationToPerform(0),

        // Kernel parameters
        alpha(1.0),
        beta1(1.0),
        beta0(0.0){

      // Interpolation
      d_interpolationMatrix = nullptr;

      // System
      B      = nullptr;
      K      = nullptr;
      Gamma  = nullptr;
      Lambda = nullptr;
      ForcePattern     = nullptr;
      d_ExcitationsSet = nullptr;

      // RK4
      d_QinitCond = nullptr;
      d_Q = nullptr;

      d_mi = nullptr;

      d_m1 = nullptr;
      d_m2 = nullptr;
      d_m3 = nullptr;
      d_m4 = nullptr;

      // CUDA
      streams    = nullptr;
      h_cuSPARSE = NULL;
    
      // Put on the device alpha and beta values for the cuSPARSE API
      CHECK_CUDA( hipMalloc((void**)&d_alpha, sizeof(reel)) )
      CHECK_CUDA( hipMalloc((void**)&d_beta1, sizeof(reel)) )
      CHECK_CUDA( hipMalloc((void**)&d_beta0, sizeof(reel)) )
      CHECK_CUDA( hipMemcpy(d_alpha, &alpha, sizeof(reel), hipMemcpyHostToDevice) )
      CHECK_CUDA( hipMemcpy(d_beta1, &beta1, sizeof(reel), hipMemcpyHostToDevice) )
      CHECK_CUDA( hipMemcpy(d_beta0, &beta0, sizeof(reel), hipMemcpyHostToDevice) )

      _loadExcitationsSet(excitationSet_, sampleRate_);
      setCUDA(nStreams);
    }



  /**
    * @brief Destroy the gpudriver::  gpudriver object
    * 
    */
    __GpuDriver::~__GpuDriver(){
      // Free memory      
      if(d_ExcitationsSet != nullptr){
        CHECK_CUDA( hipFree(d_ExcitationsSet) );
        d_ExcitationsSet = nullptr;
      }

      clearB();
      clearK();
      clearGamma();
      clearLambda();
      clearForcePattern();
      clearInitialConditions();
      clearInterpolationMatrix();
      clearModulationBuffer();
      clearDeviceStatesVector();

      if(streams != nullptr){
        for(uint i = 0; i < nStreams; i++){
          CHECK_CUDA( hipStreamDestroy(streams[i]) );
        }
        delete[] streams;
        streams = nullptr;
      }
    }



  /** __GpuDriver::loadExcitationsSet()
    * @brief Load the excitation set in the GPU memory
    * 
    * @param excitationSet_ 
    */
    int __GpuDriver::_loadExcitationsSet(std::vector< std::vector<double> > excitationSet_, 
                                         uint sampleRate_){

      sampleRate = sampleRate_;
      setTimesteps();


      // Check if the ExcitationsSet is already loaded
      excitationSet.clear();
      if(d_ExcitationsSet != nullptr){
        CHECK_CUDA( hipFree(d_ExcitationsSet) )
        d_ExcitationsSet = nullptr;
      }

      // Check the size of all the excitation vectors
      for(auto &excitation : excitationSet_){
        if(excitation.size() != excitationSet_[0].size()){
          std::cout << "Error : Excitations vectors are not of the same size" << std::endl;
          return 1;
        }
      }

      numberOfExcitations    = excitationSet_.size();
      lengthOfeachExcitation = excitationSet_[0].size();
      // Parse the input excitationSet_ to a 1D array
      for(auto &excitation : excitationSet_){
        for(auto &sample : excitation){
          excitationSet.push_back((reel)sample);
        }
      }

      // Allocate memory on the GPU
      CHECK_CUDA( hipMalloc((void**)&d_ExcitationsSet, excitationSet.size()*sizeof(reel)) )
      // Copy the ExcitationsSet to the GPU
      CHECK_CUDA( hipMemcpy(d_ExcitationsSet, excitationSet.data(), excitationSet.size()*sizeof(reel), hipMemcpyHostToDevice) )
      std::cout << "Loaded " << numberOfExcitations << " excitations of length " << lengthOfeachExcitation << " each." << std::endl;
    
      return 0;
    }



  void __GpuDriver::_setB(std::vector< matrix > & B_){
    clearB();

    B = new COOMatrix(B_);
  }

  void __GpuDriver::_setK(std::vector< matrix > & K_){
    clearK();

    K = new COOMatrix(K_);
  }

  void __GpuDriver::_setGamma(std::vector< tensor3d > & Gamma_){
    clearGamma();

    Gamma = new COOTensor3D(Gamma_);
  }

  void __GpuDriver::_setLambda(std::vector< tensor4d > & Lambda_){
    clearLambda();

    Lambda = new COOTensor4D(Lambda_);
  }

  void __GpuDriver::_setForcePattern(std::vector< std::vector<reel> > & ForcePattern_){
    clearForcePattern();

    ForcePattern = new COOVector(ForcePattern_);
  }

  void __GpuDriver::_setInitialConditions(std::vector< std::vector<reel> > & InitialConditions_){
    clearInitialConditions();

    // Initialize the number of DOF at the original size of the system
    n_dofs = InitialConditions_[0].size();

    // Allocate the QinitCond vector with the set of initials conditions
    for(size_t k(0); k<InitialConditions_.size(); k++){
      for(size_t i(0); i<InitialConditions_[k].size(); i++){
        QinitCond.push_back(InitialConditions_[k][i]);
      }
    }
  }

  void __GpuDriver::_setInterpolationMatrix(std::vector<reel> & interpolationMatrix_,
                                            uint interpolationWindowSize_){
    interpolationMatrix = interpolationMatrix_;
    interpolationWindowSize  = interpolationWindowSize_;
    interpolationNumberOfPoints = interpolationMatrix_.size()/interpolationWindowSize_;

    setTimesteps();

    // Allocate the interpolation matrix on the GPU
    CHECK_CUDA( hipMalloc((void**)&d_interpolationMatrix, interpolationMatrix.size()*sizeof(reel)) )
    // Copy the interpolation matrix to the GPU
    CHECK_CUDA( hipMemcpy(d_interpolationMatrix, interpolationMatrix.data(), interpolationMatrix.size()*sizeof(reel), hipMemcpyHostToDevice) )
  }



  void __GpuDriver::_setModulationBuffer(std::vector<reel> & modulationBuffer_){
    modulationBuffer = modulationBuffer_;
    modulationBufferSize = modulationBuffer_.size();

    // Allocate the modulation buffer on the GPU
    CHECK_CUDA( hipMalloc((void**)&d_modulationBuffer, modulationBufferSize*sizeof(reel)) )
    // Copy the modulation buffer to the GPU
    CHECK_CUDA( hipMemcpy(d_modulationBuffer, modulationBuffer.data(), modulationBufferSize*sizeof(reel), hipMemcpyHostToDevice) )
  }



  /** __GpuDriver::driver_getAmplitudes()
   * @brief 
   * 
   * @return std::vector<reel>
   */
   std::vector<reel> __GpuDriver::_getAmplitudes(bool displayComputeInfos_, bool displaySystem_){
      
    optimizeIntraStrmParallelisme();

    if(displaySystem_){
      displayAssembledSystem();
    }
    if(displayComputeInfos_){
      displayComputationInfos();
    }
    if(true){
      std::cout << "The number of steps of the simulations are" << std::endl;
      reel duration = baseNumsteps*(interpolationNumberOfPoints+1)*h;
      std::cout << "  duration = " << duration << "s" << std::endl;
      /* std::cout << "  baseNumsteps = " << baseNumsteps << std::endl;
      std::cout << "  interpolationNumberOfPoints = " << interpolationNumberOfPoints << std::endl;
      std::cout << "  total number of steps = " << baseNumsteps*(interpolationNumberOfPoints+1) << std::endl; */
      std::cout << "The timestep of the simulations are" << std::endl;
      std::cout << "  h = " << h << std::endl;
      std::cout << "  h2 = " << h2 << std::endl;
      std::cout << "  h6 = " << h6 << std::endl;
    }
    

    // Allocate the memory for the states and RK4 vectors coefficients,
    // and create the dense vector descriptors
    CHECK_CUDA( hipMalloc((void**)&d_QinitCond, n_dofs*sizeof(reel)) )
    CHECK_CUDA( hipMalloc((void**)&d_Q, n_dofs*sizeof(reel)) )
    CHECK_CUDA( hipMemcpy(d_QinitCond, QinitCond.data(), n_dofs*sizeof(reel), hipMemcpyHostToDevice) )
    // Copy the device QinitCond initial conditions vector to Q device vector
    CHECK_CUDA( hipMemcpy(d_Q, d_QinitCond, n_dofs*sizeof(reel), hipMemcpyDeviceToDevice) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_Q_desc, n_dofs, d_Q, HIP_R_32F) )

    CHECK_CUDA( hipMalloc((void**)&d_mi, n_dofs*sizeof(reel)) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_mi_desc, n_dofs, d_mi, HIP_R_32F) )
    
    CHECK_CUDA( hipMalloc((void**)&d_m1, n_dofs*sizeof(reel)) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_m1_desc, n_dofs, d_m1, HIP_R_32F) )
    CHECK_CUDA( hipMalloc((void**)&d_m2, n_dofs*sizeof(reel)) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_m2_desc, n_dofs, d_m2, HIP_R_32F) )
    CHECK_CUDA( hipMalloc((void**)&d_m3, n_dofs*sizeof(reel)) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_m3_desc, n_dofs, d_m3, HIP_R_32F) )
    CHECK_CUDA( hipMalloc((void**)&d_m4, n_dofs*sizeof(reel)) )
    CHECK_CUSPARSE( hipsparseCreateDnVec(&d_m4_desc, n_dofs, d_m4, HIP_R_32F) )



    // Allocate the matrices and vectors on the GPU
    B->AllocateOnGPU(h_cuSPARSE, d_mi_desc, d_Q_desc);
    K->AllocateOnGPU(h_cuSPARSE, d_mi_desc, d_Q_desc);
    Gamma->AllocateOnGPU();
    Lambda->AllocateOnGPU();
    ForcePattern->AllocateOnGPU();


    std::vector<reel> resultsQ;
    resultsQ.resize(n_dofs*numberOfSimulationToPerform);


    auto begin = std::chrono::high_resolution_clock::now();

    // Perform the simulations
    for(size_t k(0); k<numberOfSimulationToPerform; ++k){

      uint m(0); // Modulation index

      // Performe the rk4 steps
      for(uint t(0); t<baseNumsteps ; ++t){
        // Always performe one step without interpolation, and then performe the
        // interpolation steps
        for(uint i(0); i<=interpolationNumberOfPoints; ++i){
          rkStep(k, t, i, m);
          
          ++m;
          if(m == modulationBufferSize){
            m = 0;
          }
        }

      }



      // Copy the results of the performed simulation from the GPU to the CPU
      CHECK_CUDA( hipMemcpy(resultsQ.data()+k*n_dofs, d_Q, n_dofs*sizeof(reel), hipMemcpyDeviceToHost) )
      CHECK_CUDA( hipDeviceSynchronize() )

      // Reset Q1 and Q2 to initials conditions
      CHECK_CUDA( hipMemcpy(d_Q, d_QinitCond, n_dofs*sizeof(reel), hipMemcpyDeviceToDevice) )

      // Reset all of the other vectors to 0
      CHECK_CUDA( hipMemset(d_mi, 0, n_dofs*sizeof(reel)) )

      CHECK_CUDA( hipMemset(d_m1, 0, n_dofs*sizeof(reel)) )
      CHECK_CUDA( hipMemset(d_m2, 0, n_dofs*sizeof(reel)) )
      CHECK_CUDA( hipMemset(d_m3, 0, n_dofs*sizeof(reel)) )
      CHECK_CUDA( hipMemset(d_m4, 0, n_dofs*sizeof(reel)) )
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-begin;
    std::cout << "CUDA solver execution time: " << elapsed_seconds.count() << "s" << std::endl;

    // Cut the results vector to the correct size
    if(exceedingSimulations != 0){
      resultsQ.resize(n_dofs*(numberOfSimulationToPerform-1)+exceedingSimulations);
    }

    return std::vector<reel>{resultsQ};
   }



/****************************************************
 *              Private functions                   *
 ****************************************************/
  /** __GpuDriver::setCUDA()
    * @brief Set the parameters of the computation
    * 
    * @param nStreams_ 
    * @param nIntraStrmParallelism_ 
    * @return int 
    */
    int __GpuDriver::setCUDA(uint nStreams_){
      nStreams = nStreams_;

      /* // Query the number of available devices
      int nDevices;
      CHECK_CUDA( hipGetDeviceCount(&nDevices) ) */

      // Query the device parameters
      CHECK_CUDA( hipGetDevice(&deviceId) )
      CHECK_CUDA( hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId) )

      nThreadsPerBlock = 128;
      nBlocks = numberOfSMs * 32;

      // Spawn the streams
      streams = new hipStream_t[nStreams];
      for(uint i = 0; i < nStreams; i++){
        CHECK_CUDA( hipStreamCreate(&streams[i]) )
      }

      // Create the cuBLAS handle
      CHECK_CUBLAS( hipblasCreate(&h_cublas) )
      CHECK_CUBLAS( hipblasSetPointerMode(h_cublas, HIPBLAS_POINTER_MODE_DEVICE) )

      CHECK_CUBLAS( hipblasSetStream(h_cublas, streams[0]) )

      // Create the cuSPARSE handle
      CHECK_CUSPARSE( hipsparseCreate(&h_cuSPARSE) )
      CHECK_CUSPARSE( hipsparseSetPointerMode(h_cuSPARSE, HIPSPARSE_POINTER_MODE_DEVICE) )

      CHECK_CUSPARSE( hipsparseSetStream(h_cuSPARSE, streams[0]) )

      return 0;
    }



  void __GpuDriver::setTimesteps(){
    if(!interpolationMatrix.empty()){
      h  = 1.0/(sampleRate*(interpolationNumberOfPoints+1));
    }
    else{
      h  = 1.0/sampleRate;
    }

    h2 = h/2.0;
    h6 = h/6.0;
  }



  /**
   * @brief Compute the derivatives of the system
   * 
   */
   inline void __GpuDriver::derivatives(hipsparseDnVecDescr_t m_desc, 
                                        hipsparseDnVecDescr_t q_desc, 
                                        uint k, 
                                        uint t,
                                        uint i,
                                        uint m){

    // Get the pointers from the descriptors
    reel *pm; reel *pq;
    CHECK_CUSPARSE( hipsparseDnVecGetValues(m_desc, (void**)&pm) )
    CHECK_CUSPARSE( hipsparseDnVecGetValues(q_desc, (void**)&pq) )
    
    // k = B.d_ki + K.d_mi + Gamma.d_mi² + Lambda.d_mi³ + ForcePattern.d_ExcitationsSet
    // k = B.d_ki
    hipsparseSpMV(h_cuSPARSE, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                 d_alpha, 
                 B->sparseMat_desc, 
                 q_desc,
                 d_beta0, 
                 m_desc, 
                 HIP_R_32F, 
                 HIPSPARSE_SPMV_ALG_DEFAULT, 
                 B->d_buffer);
    
    // k += K.d_mi
    hipsparseSpMV(h_cuSPARSE, 
                 HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                 d_alpha, 
                 K->sparseMat_desc, 
                 q_desc, 
                 d_beta1, 
                 m_desc, 
                 HIP_R_32F, 
                 HIPSPARSE_SPMV_ALG_DEFAULT, 
                 K->d_buffer);
    
    // k += Gamma.d_mi²
    SpT3dV<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(Gamma->d_val,
                                                         Gamma->d_row, 
                                                         Gamma->d_col,
                                                         Gamma->d_slice, 
                                                         Gamma->nzz, 
                                                         pq, 
                                                         pm);
    
    // k += Lambda.d_mi³
    SpT4dV<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(Lambda->d_val,
                                                         Lambda->d_row, 
                                                         Lambda->d_col,
                                                         Lambda->d_slice, 
                                                         Lambda->d_hyperslice,
                                                         Lambda->nzz, 
                                                         pq, 
                                                         pm);
    
    // Conditional release of the excitation in the case of a simulation longer 
    // than the excitation length
    if(t < lengthOfeachExcitation){
      // k += ForcePattern.d_ExcitationsSet
      modterpolator(pm, k, t, i, m);
    }
   }

  

  /**
   * @brief Performe a single Runge-Kutta step
   * 
   */
   void __GpuDriver::rkStep(uint k, 
                            uint t,
                            uint i,
                            uint m){

    // Compute the derivatives
    derivatives(d_m1_desc, d_Q_desc, k, t, i, m);

      updateSlope<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(d_mi, d_Q, d_m1, h2, n_dofs);

    derivatives(d_m2_desc, d_mi_desc, k, t, i+1, m);

      updateSlope<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(d_mi, d_Q, d_m2, h2, n_dofs);

    derivatives(d_m3_desc, d_mi_desc, k, t, i+1, m);

      updateSlope<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(d_mi, d_Q, d_m3, h, n_dofs);

    derivatives(d_m4_desc, d_mi_desc, k, t, i+2, m);

    // Compute next state vector Q
    integrate<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>(d_Q, d_m1, d_m2, d_m3, d_m4, h6, n_dofs);
   

    // Store the result in the trajectory
    reel temp = 0.;
    hipMemcpy(&temp, d_Q, sizeof(reel), hipMemcpyDeviceToHost);

    h_trajectory.push_back(temp);
   }



  inline void __GpuDriver::modterpolator(reel* Y,
                                         uint  k,
                                         uint  t,
                                         uint  i,
                                         uint  m){

    // "currentSimulation" refers to the simulation number in the case of
    // wich multiple simulation are needed to compute the system against all
    // of the excitation file
    uint currentSimulation = k/intraStrmParallelism;
    uint systemStride      = n_dofs/intraStrmParallelism;
    uint adjustedTime          = t;
    uint adjustedInterpolation = i;

    uint useCase = 0;

    if(interpolationNumberOfPoints == 0){
      adjustedTime         += i;
      adjustedInterpolation = 0;

      useCase = 0;
    }
    else{
      if(i > interpolationNumberOfPoints){
        adjustedTime          += 1;
        adjustedInterpolation -= (interpolationNumberOfPoints+1);
      }
      
      if(adjustedInterpolation == 0){
        useCase = 0;
      }
      else{
        useCase = 1;
      }
    }

    switch(useCase){
      case 0: // Just apply the force
        applyForces<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>
                                              (ForcePattern->d_val, 
                                              ForcePattern->d_indice, 
                                              ForcePattern->nzz, 
                                              d_ExcitationsSet,
                                              lengthOfeachExcitation, 
                                              currentSimulation,
                                              systemStride,
                                              Y, 
                                              adjustedTime,
                                              d_modulationBuffer,
                                              m);
        break;
      case 1: // Interpolate the force
        interpolateForces<<<nBlocks, nThreadsPerBlock, 0, streams[0]>>>
                                                    (ForcePattern->d_val, 
                                                    ForcePattern->d_indice, 
                                                    ForcePattern->nzz, 
                                                    d_ExcitationsSet,
                                                    lengthOfeachExcitation, 
                                                    currentSimulation,
                                                    systemStride,
                                                    Y, 
                                                    adjustedTime,
                                                    d_interpolationMatrix,
                                                    interpolationWindowSize,
                                                    adjustedInterpolation,
                                                    d_modulationBuffer,
                                                    m);
        break;
    }
  }



  inline void __GpuDriver::displayAssembledSystem(){
    std::cout << "Here is the assembled system" << std::endl;
    std::cout << "B:" << std::endl << *B << std::endl;
    std::cout << "K:" << std::endl << *K << std::endl;
    std::cout << "Gamma:" << std::endl << *Gamma << std::endl;
    std::cout << "Lambda:" << std::endl << *Lambda << std::endl;
    std::cout << "ForcePattern:" << std::endl << *ForcePattern << std::endl;
    std::cout << "QinitCond:" << std::endl; printVector(QinitCond);

    std::cout << "InterpolationMatrix:" << std::endl;
    if(!interpolationMatrix.empty()){
      for(uint i=0; i<interpolationNumberOfPoints; ++i){
        for(uint j=0; j<interpolationWindowSize; ++j){
          std::cout << interpolationMatrix[i*interpolationWindowSize + j] << " ";
        }
        std::cout << std::endl;
      }
    }
    else{
      std::cout << "No interpolation matrix has been provided" << std::endl;
    }
    std::cout << std::endl;

    std::cout << "Modulation buffer:" <<  std::endl;
    printVector(modulationBuffer);

  }


  
  inline void __GpuDriver::displayComputationInfos(){
    std::cout << "A system with " << n_dofs << " DOFs has been assembled" << std::endl;
    std::cout << "  This system is composed of " << intraStrmParallelism << " parallelized simulations of " << n_dofs/intraStrmParallelism << " DOF each." << std::endl;
    std::cout << "  The total number of excitation files is " << numberOfExcitations << std::endl;
    std::cout << "  Hence the number of simulation to perform is " << numberOfSimulationToPerform << std::endl;
    if(baseNumsteps < lengthOfeachExcitation){
      std::cout << "  Warning: The number of steps to perform is inferior to the length of the excitation files" << std::endl;
    }
  }



  /** __GpuDriver::checkAndDestroy()
   * @brief Check the device pointer array and destroy them if they are not null
   * 
   */
   void __GpuDriver::clearDeviceStatesVector(){
    
    if(d_Q != nullptr){
      CHECK_CUDA( hipFree(d_Q) )
      d_Q = nullptr;
    }

    if(d_mi != nullptr){
      CHECK_CUDA( hipFree(d_mi) )
      d_mi = nullptr;
    }

    if(d_m1 != nullptr){
      CHECK_CUDA( hipFree(d_m1) )
      d_m1 = nullptr;
    }
    if(d_m2 != nullptr){
      CHECK_CUDA( hipFree(d_m2) )
      d_m2 = nullptr;
    }
    if(d_m3 != nullptr){
      CHECK_CUDA( hipFree(d_m3) )
      d_m3 = nullptr;
    }
    if(d_m4 != nullptr){
      CHECK_CUDA( hipFree(d_m4) )
      d_m4 = nullptr;
    }
   }

  void __GpuDriver::clearB(){
    if(B != nullptr){
      delete B;
      B = nullptr;
    }
  }

  void __GpuDriver::clearK(){
    if(K != nullptr){
      delete K;
      K = nullptr;
    }
  }

  void __GpuDriver::clearGamma(){
    if(Gamma != nullptr){
      delete Gamma;
      Gamma = nullptr;
    }
  }

  void __GpuDriver::clearLambda(){
    if(Lambda != nullptr){
      delete Lambda;
      Lambda = nullptr;
    }
  }

  void __GpuDriver::clearForcePattern(){
    if(ForcePattern != nullptr){
      delete ForcePattern;
      ForcePattern = nullptr;
    }
  }

  void __GpuDriver::clearInitialConditions(){
    if(QinitCond.size() != 0){
      QinitCond.clear();
    }
    if(d_QinitCond != nullptr){
      CHECK_CUDA( hipFree(d_QinitCond) )
      d_QinitCond = nullptr;
    }
  }

  void __GpuDriver::clearInterpolationMatrix(){
    if(interpolationMatrix.size() != 0){
      interpolationMatrix.clear();
    }
    if(d_interpolationMatrix != nullptr){
      CHECK_CUDA( hipFree(d_interpolationMatrix) )
      d_interpolationMatrix = nullptr;
    }
  }

  void __GpuDriver::clearModulationBuffer(){
    if(modulationBuffer.size() != 0){
      modulationBuffer.clear();
    }
    if(d_modulationBuffer != nullptr){
      CHECK_CUDA( hipFree(d_modulationBuffer) )
      d_modulationBuffer = nullptr;
    }
  }



/**
 * @brief Optimize the parallelism of the kernel
 * 
 */
  void __GpuDriver::optimizeIntraStrmParallelisme(){
    
    // 1. Get free storage on the GPU
    size_t freeSpace, totalSpace;
    CHECK_CUDA( hipMemGetInfo(&freeSpace, &totalSpace) )

    /* std::cout << "Free space on the GPU: " << freeSpace << " bytes" << std::endl;
    std::cout << "Total space on the GPU: " << totalSpace << " bytes" << std::endl;
    std::cout << "Used space " << totalSpace-freeSpace << " bytes" << std::endl; */

    // 2. Compute the size required by 1 instance of the system

    // .1 Size of the matrix of the system
    size_t sizeOfSystem(0);
    sizeOfSystem += B->memFootprint();
    sizeOfSystem += K->memFootprint();
    sizeOfSystem += Gamma->memFootprint();
    sizeOfSystem += Lambda->memFootprint();
    sizeOfSystem += ForcePattern->memFootprint();

    // std::cout << "Size of 1 system: " << sizeOfSystem << " bytes" << std::endl;

    // .2 Size of the rk4 and states vector needed for the computation
    size_t sizeOfStates(0);
    sizeOfStates += 13*sizeof(reel)*n_dofs;

    // std::cout << "Size of the states: " << sizeOfStates << " bytes" << std::endl;

    size_t totalSize = sizeOfSystem + sizeOfStates;

    // 3. Compute the max number of system that we can fit in the gpu memory

    size_t maxNumberOfSystem = (0.8*freeSpace) / totalSize;

    if(maxNumberOfSystem > numberOfExcitations){
      maxNumberOfSystem = numberOfExcitations;
    }

    intraStrmParallelism = maxNumberOfSystem;

    numberOfSimulationToPerform = numberOfExcitations / intraStrmParallelism;
    exceedingSimulations = numberOfExcitations % intraStrmParallelism;
    if(exceedingSimulations != 0){
      numberOfSimulationToPerform++;
    }

    // Extend each system by the number of intra-stream parallelization wanted
    std::array<uint, 6> dofChecking = {B->ExtendTheSystem(intraStrmParallelism-1), 
                                       K->ExtendTheSystem(intraStrmParallelism-1), 
                                       Gamma->ExtendTheSystem(intraStrmParallelism-1), 
                                       Lambda->ExtendTheSystem(intraStrmParallelism-1), 
                                       ForcePattern->ExtendTheSystem(intraStrmParallelism-1),
                                       extendTheVector(QinitCond, intraStrmParallelism-1)};

    // Checking that each system is of the same size
    for(uint i = 0; i < dofChecking.size(); i++){
      if(dofChecking[i] != dofChecking[0]){
        std::cout << "Error : The number of DOFs is not the same for all the Matrix describing the system" << std::endl;
      }
    }

    // Modify if needed the number of DOFs
    if(n_dofs != dofChecking[0]){
      n_dofs = dofChecking[0];
    }
  }




