#include "hip/hip_runtime.h"
/**
 * @file kernels.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2022-12-02
 * 
 * @copyright Copyright (c) 2022
 * 
 */


#include "kernels.cuh"



/** customSpTd2V()
 * @brief Perform the coo sparse tensor - dense vector square multiplication
 * 
 */
 __global__
 void customSpTd2V(reel *d_val, 
                   uint *d_row, 
                   uint *d_col, 
                   uint *d_slice, 
                   uint nzz,
                   reel* X, 
                   reel* Y){

  uint index  = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride = blockDim.x * gridDim.x;  

  for(uint k = index; k < nzz; k += stride){
    atomicAdd(&Y[d_slice[k]], d_val[k] * X[d_row[k]] * X[d_col[k]]);
  }
 }



 /** customSpTd3V()
 * @brief Perform the coo sparse tensor 4d - dense vector multiplication (order 3)
 * 
 */
 __global__
 void customSpTd3V(reel *d_val, 
                   uint *d_row, 
                   uint *d_col, 
                   uint *d_slice, 
                   uint *d_hyperslice,
                   uint nzz,
                   reel* X, 
                   reel* Y){

  uint index  = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride = blockDim.x * gridDim.x;  

  for(uint k = index; k < nzz; k += stride){
    atomicAdd(&Y[d_hyperslice[k]], d_val[k] * X[d_row[k]] * X[d_col[k]] * X[d_slice[k]]);
  }
 }



/** customAxpbyMultiForces()
 * @brief Performe a custom Axpby operation on the forces vector to accomodate multi excitation file
 * parallelisme acrose a single system
 * 
 */
 __global__
 void customAxpbyMultiForces(reel* d_val, 
                             uint* d_indice, 
                             uint nzz, 
                             reel* excitationsSet,
                             uint lengthOfeachExcitation, 
                             uint kSim, 
                             reel* Y, 
                             uint n, 
                             uint t,
                             uint intraStrmParallelism){

  uint dofStride = n/intraStrmParallelism;
  uint selectedExcitation = kSim*intraStrmParallelism;

  uint index  = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride = blockDim.x * gridDim.x;  

  for(uint k = index; k<nzz; k += stride){
    // Y[d_indice[k]] += 0.0;
    Y[d_indice[k]] += d_val[k]*excitationsSet[(selectedExcitation+d_indice[k]/dofStride)*lengthOfeachExcitation + t];

  }
 }



/** updateSlope()
 * @brief Compute the next estimation vectors
 * 
 */
 __global__
 void updateSlope(reel* rki, 
                  reel* q, 
                  reel* rk, 
                  reel dt, 
                  uint n){

  uint index  = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride = blockDim.x * gridDim.x;  

  for(uint k = index; k < n; k += stride){
    rki[k] = q[k] + dt*rk[k];
  }
 }



/** integrate()
 * @brief Compute the next state vector based on the rk4 estimations
 * 
 */
 __global__
 void integrate(reel* q, 
                reel* rk1, 
                reel* rk2, 
                reel* rk3, 
                reel* rk4, 
                reel h6, 
                uint n){

  uint index  = threadIdx.x + blockIdx.x * blockDim.x;
  uint stride = blockDim.x * gridDim.x;  

  for(uint k = index; k < n; k += stride){
    q[k] += h6*(rk1[k] + 2*rk2[k] + 2*rk3[k] + rk4[k]);
  }
 }
